#include "hip/hip_runtime.h"


#include <cstdio>
#include <hip/hip_complex.h>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <cstdlib>
#include <cstring>
#define DEBUG
using namespace std;
const int N = 2e5 + 10;
int t, n;
__constant__ int T[1];

inline hipError_t checkCuda(hipError_t result){
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__  void vector_mul(hipfftDoubleComplex *a, hipfftDoubleComplex *b){
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < T[0]; i += numThreads) {//why can't i use *T in here
		hipDoubleComplex c = hipCmul(a[i], b[i]);
		a[i] = make_hipDoubleComplex(hipCreal(c) / T[0], hipCimag(c) / T[0]);
	}
}

__global__ void get_ans(int *ans, hipfftDoubleComplex *a) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < T[0]; i += numThreads)//why can't i use *T in here
		ans[i] = (int)(hipCreal(a[i]) + 0.5);

}

int main(){
	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, 0));
	//timing start
	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//intialize
	FILE *in=fopen("fft.in","r"), *out=fopen("cufft.out","w");
	fscanf(in, "%d", &n);
	t = 1; while (t < n + n) t <<= 1;
	//memory allocation
	int tt[1]; tt[0] = t;
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(T), tt, sizeof(int)));
	int size = sizeof(hipfftDoubleComplex)*t, size2 = sizeof(int)*t;
	int* h_ans = (int*)calloc(t, sizeof(int));
	char* str = (char*)malloc(sizeof(char)*(n + 10));
	int* ans;
	checkCuda(hipMalloc((void **)&ans, size2));
	hipfftDoubleComplex *a, *b;
	hipfftDoubleComplex *h_a = (hipfftDoubleComplex *)calloc(t, sizeof(hipfftDoubleComplex));
	hipfftDoubleComplex *h_b = (hipfftDoubleComplex *)calloc(t, sizeof(hipfftDoubleComplex));
	checkCuda(hipMalloc((void **)&a, size));
	checkCuda(hipMalloc((void **)&b, size));
	//input and memcpy
	fscanf(in, "%s", str); for (int i = 0; i < n; i++) h_a[i] = make_hipDoubleComplex((double)str[n - i - 1] - '0', 0.0);
	fscanf(in, "%s", str); for (int i = 0; i < n; i++) h_b[i] = make_hipDoubleComplex((double)str[n - i - 1] - '0', 0.0);
	checkCuda(hipMemcpy(a, h_a, size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(b, h_b, size, hipMemcpyHostToDevice));
	//dft
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, t, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "cufft plan create failed!");
		return 1;
	}
	if (hipfftExecZ2Z(plan, a, a, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: a ExecZ2Z Forward failed");
		return 2;
	}
	if (hipfftExecZ2Z(plan, b, b, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: b ExecZ2Z Forward failed");
		return 2;
	}
	//multiply
	vector_mul<<<t / prop.maxThreadsPerBlock + 1, prop.maxThreadsPerBlock>>>(a, b);
	checkCuda(hipDeviceSynchronize());
	//idft
	if (hipfftExecZ2Z(plan, a, a, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: a ExecZ2Z Inverse failed");
		return 4;
	}
	if (hipfftDestroy(plan) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: fft plan destroy failed");
		return 5;
	}
	//change into integer: serial or parallel??
	checkCuda(hipMemcpy(h_a, a, size, hipMemcpyDeviceToHost));
	//for (int i = 0; i<t; i++) ans[i] = (int)(cuCreal(h_a[i]) + 0.5);
	//for (int i = 0; i<t; i++) ans[i + 1] += ans[i] / 10, ans[i] %= 10;
	get_ans<<<t / prop.maxThreadsPerBlock + 1, prop.maxThreadsPerBlock >>>(ans, a);
	checkCuda(hipMemcpy(h_ans, ans, size2, hipMemcpyDeviceToHost));
	for (int i = 0; i<t; i++) h_ans[i + 1] += h_ans[i] / 10, h_ans[i] %= 10;
	//output
	while (!h_ans[t-1]) t--;
	for (int i = t-1; i >= 0; i--) fprintf(out, "%d", h_ans[i]);
	//delete
	checkCuda(hipFree(a));
	checkCuda(hipFree(b));
	checkCuda(hipFree(ans));
	free(h_ans);
	free(h_a);
	free(h_b);
	free(str);
	//timing end
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%lf\t", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	if (checkCuda(hipDeviceReset()) != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 10;
	}
	return 0;
}